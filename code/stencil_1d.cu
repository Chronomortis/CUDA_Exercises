
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <algorithm>

using namespace std;

#define N 4096
#define RADIUS 3
#define BLOCK_SIZE 16

__global__ void stencil_1d(int *in, int *out) {
    __shared__ int temp[BLOCK_SIZE + 2 * RADIUS];
    int gindex = threadIdx.x + blockIdx.x * blockDim.x;
    int lindex = threadIdx.x + RADIUS; // Error?

    // Read input elements into shared memory
    temp[lindex] = in[gindex];
    if (threadIdx.x < RADIUS) {
      temp[lindex - RADIUS] = in[gindex - RADIUS];
      temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE];
    }

    // Synchronize (ensure all the data is available)
    __syncthreads();
	
	// Apply the stencil
    int result = 0;
    for (int offset = -RADIUS; offset <= RADIUS; offset++) 
	{
		int count = 1;
		result += temp[lindex + offset];
		if(gindex == 0) 
		{
			printf("Iteration Count: %d, %d\n",count,temp[lindex + offset]);
			
		} 
	}
	
    // Store the result
    out[gindex] = result;
	if(gindex == 0) printf("Result: %d\n",result);
}

void fill_ints(int *x, int n) {
  fill_n(x, n, 1);
}

int main(void) {
  int *in, *out; // host copies of a, b, c
  int *d_in, *d_out; // device copies of a, b, c

  // Alloc space for host copies and setup values
  int size = (N + 2 * RADIUS) * sizeof(int);  // I'm not sure about that!!
  in = (int *)malloc(size); fill_ints(in, N + 2*RADIUS);
  out = (int *)malloc(size); fill_ints(out, N + 2*RADIUS);

  // Alloc space for device copies
  hipMalloc((void **)&d_in, size);
  hipMalloc((void **)&d_out, size);

  // Copy to device
  hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);
  hipMemcpy(d_out, out, size, hipMemcpyHostToDevice);

  // Launch stencil_1d() kernel on GPU
  stencil_1d<<<N/BLOCK_SIZE , BLOCK_SIZE>>>(d_in + RADIUS, d_out + RADIUS);   // Only mistake was here...

  // Copy result back to host
  hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);

  // Error Checking
  int error_count = 0;
  for (int i = 0; i < N + 2*RADIUS; i++) {
    if (i<RADIUS || i>=N+RADIUS){
      if (out[i] != 1) {
		printf("Mismatch at index %d, was: %d, should be: %d\n", i, out[i], 1);
    	error_count++;
	  }
	} else {
      if (out[i] != 1 + 2*RADIUS) 
	  {
		printf("Mismatch at index %d, was: %d, should be: %d\n", i, out[i], 1 + 2*RADIUS);
    	error_count++;
	  }
	}
  }
  printf("%d\n",error_count++);


  // Cleanup
  free(in); free(out);
  hipFree(d_in); hipFree(d_out);
  printf("Success!\n");
  return 0;
}
