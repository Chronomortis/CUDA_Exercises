
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <stdio.h>
#include <algorithm>
#define M 4
#define N 4

__global__ void matrix_multiplication(int *a,int *b, int *c,int divisor ,int row, int column,int n) 
{
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	if(index < n) 
	{
	
		int res_row = index / column;
		int res_column = index % column;
		for(int i = 0; i < divisor; i++) 
		{
			c[index] += a[res_row * divisor + i] * b[column * i + res_column];
		}
		printf("C Index %i Value: %i\n",index,c[index]);
	}
}




int main() 
{
int *a,*b,*c; // host pointers
int *d_a,*d_b,*d_c; //device pointers.
int divisor,count = 1;

// Finding the greatest common denominator.
while(count < sqrt(M) && count < sqrt(N)) 
{
	if(M % count == 0 && N % count == 0) 
	{
		divisor = count;
	}	
	count++;
}
int size_c = (M / divisor) * (N / divisor);
printf("Size C Value: %i\n Divisor Value: %i\n",size_c,divisor);
a = (int *) malloc(N * sizeof(int));
b = (int *) malloc(M * sizeof(int));
c = (int *) malloc(size_c * sizeof(int));

for(int i = 0; i < M; i++) 
{
	b[i] = i + 2;
	std::cout << b[i] << "a member of matrix B" << std::endl;
}
for(int j = 0; j < N; j++) 
{
	a[j] = j + 1;
	std::cout << a[j] << "a member of matrix A" << std::endl;
}

hipMalloc((void **)&d_a, N * sizeof(int));
hipMalloc((void **)&d_b, M * sizeof(int));
hipMalloc((void **)&d_c, size_c * sizeof(int));

hipMemcpy(d_a,a, N * sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(d_b,b, M * sizeof(int),hipMemcpyHostToDevice);


matrix_multiplication<<<( ((M + N - 1) / 32) + 1),32>>>(d_a,d_b,d_c,divisor,(N / divisor),(M / divisor),size_c);
hipMemcpy(c,d_c, size_c * sizeof(int), hipMemcpyDeviceToHost);

hipDeviceSynchronize();
for(int i = 0; i < size_c; i++) 
{
	if(i % (N / divisor) == 0 && i != 0) {std::cout << std::endl;}
	std::cout << c[i] << " ";
}
std::cout << "\n";
free(a);
free(b);
free(c);

hipFree(d_a);
hipFree(d_b);
hipFree(d_c);
return 0;




}
