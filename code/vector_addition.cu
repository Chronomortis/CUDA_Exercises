#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;

#define N 512
/*CUDA Intro:
Host = CPU
Device = GPU
The var. within those can't (normally) be dereferenced or passed to the other

To handle GPU/device memory: hipFree(),hipMemcpy() and hipMalloc() can be used.

Kernel Launch Configurations: 

 */

__global__ void add(int *a, int *b, int *c) // Basically a function that is executed from the GPU.
{
	// blockIdx.x works because block size is 1 and amount of threads are also 1 with a thread ID of 0.
	c[blockIdx.x] == a[blockIdx.x] + b[blockIdx.x];

}

int main(void) // I don't know about the void part. 
{
	int *a, *b, *c; // Pointers for host
	int *d_a,*d_b,*d_c; // Pointers for the device.
	int size = N * sizeof(int);
	
	// Allocate memory space on GPU for the vectors. Don't know about void ** part.
	
	hipMalloc((void **) &d_a, size);
	hipMalloc((void **) &d_b, size);
	hipMalloc((void **) &d_c, size);

	
	// Allocating memory on CPU and initializing a & b vectors with random numbers with the amount of N.
	a = (int *)malloc(size);
	b = (int *)malloc(size);
	c = (int *)malloc(size);
	random_ints(a,N);
	random_ints(b,N);

	//Copying data from the host(CPU) to the device(GPU).
	hipMemcpy(d_a, a, size, cudaMemcpyHosttoDevice);
	hipMemcpy(d_b, b, size, cudaMemcpyHosttoDevice); // Parameters for hipMemcpy = GPU pointer, CPU pointer, size of the copying, the direction of the copying.
	
	// Launching the add() kernel from the GPU. Kernel takes in var. from the GPU. 
	add<<<N,1>>>(d_a,d_b,d_c) 
	
	// Copy results back to the CPU.
	hipMemcpy(c,d_c,size,cudaMemcpyDevicetoHost);


	// Cleanup:
	free(a);
	free(b);
	free(c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;






}
